#include "hip/hip_runtime.h"
#include "class.hpp"
#include <cmath>
#include "../common/book.h"
#define threadsPerBlock 16

using namespace std;

float frand_a_b(float a, float b)
{
    return ( rand()/(float)RAND_MAX ) * (b-a) + a;
}

__global__ void dot( float *a, float *b, float *c, int *N, float *add) {
	 __shared__ float cache[threadsPerBlock];
	 int tid = threadIdx.x + blockIdx.x * blockDim.x;
	 int cacheIndex = threadIdx.x;
	 float temp = 0;
	 while (tid < *N) {
		b[tid] += *add * a[tid];
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	 }
	 cache[cacheIndex] = temp;
	 
	 __syncthreads();
	 
	 int i = blockDim.x/2;
	 while (i != 0) {
		 if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		 __syncthreads();
		 i /= 2;
	 }
	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

__global__ float para_learning(float* input, float error_factor, float* val_test, float* m_weight_old, float* m_weight, int* nb_branch, float *bias){
	
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	float y = 0.f;
    float add = 0.f;
	 for(int i = 0; i<m_nb_branchs; i++)
    {
        m_weight_old[i] = m_weight[i];
    }
    y = test(input);
    add = y*(1-y)*error_factor;
    for(int i = 0; i<m_nb_branchs; i++)
    {
        m_weight[i] += add*input[i]*learn_rate;
    }
	m_bias += add*learn_rate;
	return add;	
	
}

Neur::Neur(int nb_branchs)
{
    m_nb_branchs = nb_branchs;
    m_weight = new float[nb_branchs];
    for(int i = 0; i < nb_branchs; i++)
    {
        m_weight[i] = 0.5;
    }
	m_weight_old = new float[nb_branchs];
    for(int i = 0; i < nb_branchs; i++)
    {
        m_weight_old[i] = 0.5;
    }
}

Neur::Neur()
{
    m_nb_branchs = 1;
    m_weight = new float[1];
    for(int i = 0; i < 1; i++)
    {
        m_weight[i] = frand_a_b(0.f, 1.f);
    }
	m_weight_old = new float[1];
    for(int i = 0; i < 1; i++)
    {
        m_weight_old[i] = frand_a_b(0.f, 1.f);
    }
	m_bias = 0;	
}

void Neur::setBranchs(int nb){
	
	
	float *tempo = new float[nb];
	for(int i = 0; i<nb; i++){
		if(i<m_nb_branchs)
			tempo[i] = m_weight[i];
		else
			tempo[i] = 0;
	}
	delete m_weight;
	m_weight = tempo;
	delete tempo;
	tempo = new float[nb];
	for(int i = 0; i<nb; i++){
		tempo[i] = m_weight[i];
	}
	delete m_weight_old;
	m_weight_old = tempo;
	m_nb_branchs = nb;
}

float Neur::learning(float* input, float error_factor)
{
	/* Linear neuron
	int nb_blocks = ((m_nb_branchs+(threadsPerBlock-1))/threadsPerBlock);
	
	float *c = (float *)malloc(nb_blocks*sizeof(float));
	float *dev_a, *dev_b, *dev_partial_c, *dev_add; 
	int *dev_N;	
	
	HANDLE_ERROR( hipMalloc( (void**)&dev_a, m_nb_branchs*sizeof(float) ) );
	HANDLE_ERROR( hipMemcpy( dev_a, input, m_nb_branchs*sizeof(float), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, m_nb_branchs*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_partial_c, ((m_nb_branchs+15)/16)*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_N, sizeof(int) ) );
	HANDLE_ERROR( hipMemcpy( dev_N, &m_nb_branchs, sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_add, sizeof(int) ) );
	
    float out = 0.f;
    float error = m_expect - out;
    float add = 0.f;
    while(error > m_accept || -error > m_accept)
    {
        add = error*learn_rate;
		HANDLE_ERROR( hipMemcpy( dev_add, &add, sizeof(float), hipMemcpyHostToDevice ) );
        out = 0.f;
		HANDLE_ERROR( hipMemcpy( dev_b, m_weight, m_nb_branchs*sizeof(float), hipMemcpyHostToDevice ) );
		
		dot<<<nb_blocks,threadsPerBlock>>>( dev_a, dev_b, dev_partial_c, dev_N, dev_add);
		
		HANDLE_ERROR( hipMemcpy(c, dev_partial_c, nb_blocks*sizeof(float), hipMemcpyDeviceToHost ) );
		
		for(int i = 0; i<nb_blocks; i++){
            out += c[i];
        }
		
		HANDLE_ERROR( hipMemcpy(m_weight, dev_b, m_nb_branchs*sizeof(float), hipMemcpyDeviceToHost ) );
		
		cout << "Out reel:" << out << endl;
		
        error =(m_expect - out);
        cout << "Error : " << error << endl;
    }
	/*hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_partial_c );
	hipFree( dev_add );
	hipFree( dev_N );
	free(c);*/
	 //Logistic neuron
	 for(int i = 0; i<m_nb_branchs; i++)
    {
        m_weight_old[i] = m_weight[i];
    }
    float y = 0.f;
    float add = 0.f;
    y = test(input);
    add = y*(1-y)*error_factor;
    for(int i = 0; i<m_nb_branchs; i++)
    {
        m_weight[i] += add*input[i]*learn_rate;
    }
	m_bias += add*learn_rate;
	return add;
}

float Neur::test(float *input)
{
	int nb_blocks = ((m_nb_branchs+(threadsPerBlock-1))/threadsPerBlock);
	
	float *c = (float *)malloc(nb_blocks*sizeof(float));
	float *dev_a, *dev_b, *dev_partial_c, *dev_add; 
	int *dev_N;
	
	HANDLE_ERROR( hipMalloc( (void**)&dev_a, m_nb_branchs*sizeof(float) ) );
	HANDLE_ERROR( hipMemcpy( dev_a, input, m_nb_branchs*sizeof(float), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, m_nb_branchs*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_partial_c, ((m_nb_branchs+(threadsPerBlock-1))/threadsPerBlock)*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_N, sizeof(int) ) );
	HANDLE_ERROR( hipMemcpy( dev_N, &m_nb_branchs, sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_add, sizeof(int) ) );
	
    float out = 0.f;
    float add = 0.f;
	HANDLE_ERROR( hipMemcpy( dev_add, &add, sizeof(float), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_b, m_weight, m_nb_branchs*sizeof(float), hipMemcpyHostToDevice ) );
		
	dot<<<nb_blocks,threadsPerBlock>>>( dev_a, dev_b, dev_partial_c, dev_N, dev_add);
		
	HANDLE_ERROR( hipMemcpy(c, dev_partial_c, nb_blocks*sizeof(float), hipMemcpyDeviceToHost ) );
		
	for(int i = 0; i<nb_blocks; i++){
        out += c[i];
    }

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_partial_c );
	hipFree( dev_add );
	hipFree( dev_N );
	free(c);
	return sigmo(out);
}

float Neur::sigmo(float val){
    return 1/(1+exp(-val));
}
