#include "hip/hip_runtime.h"
#include "class.hpp"

using namespace std;

__global__ void para_learning(float* add, float* y, float* error_factor, float* bias, float* nb_branchs, float* input, float* N, float** w, float** w_old){ // N, error_factor and input : non array
	
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	 while (tid < *N) {
		 for(int i = 0; i<m_nb_branchs; i++)
		{
			m_weight_old[i] = m_weight[i];
		}
		add[tid] = y*(1-y)*error_factor;
		for(int i = 0; i<nb_branchs[tid] ; i++)
		{
			m_weight[tid] [i] += add[tid] *input[i]*learn_rate;
		}
		m_bias += add*learn_rate;
		tid += blockDim.x * gridDim.x;
	 }	
}

Net::Net(int width, int nb_input){
	m_width = width;
	input_layer = new Neur[width];
	output_layer = new Neur(width);
	for(int i = 0; i < width; i++){
		input_layer[i].setBranchs(nb_input);
	}
	m_nb_input = nb_input;
}

float Net::learning(float m_expect, float *input){
	
	float *transition = new float[m_width];
    float output = 0.f;
    for(int  i = 0; i<m_width; i++)
    {
        transition[i]  = input_layer[i].test(input);
    }
    output = output_layer->test(transition);
    float error = m_expect - output;
    float add = output_layer->learning(transition, error);
    
	//Parallel
	float  **y, **bias, **error_factor, w, w_old;
	float **dev_y, **dev_bias, *dev_nb_branchs, **dev_error_factor,  *dev_input, *dev_N, *dev_w, *dev_w_old;
	
	y = (float*)malloc(sizeof(float*) * m_width);
	bias = (float*)malloc(sizeof(float*) * m_width);
	error_factor = (float*)malloc(sizeof(float) * m_width);
	w = (float**)malloc(sizeof(float*) * m_width);
	w_old = (float**)malloc(sizeof(float*) * m_width);
	
	for(int  i = 0; i<m_width; i++)
    {
		w[i] = (float*) malloc (sizeof(float) * m_nb_input);
		w_old[i] = (float*) malloc (sizeof(float) * m_nb_input);
    }
	
	//alloc
	HANDLE_ERROR( hipMalloc( (void***)&dev_y, m_width*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void***)&dev_bias, m_width*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_nb_branchs, sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void***)&dev_error_factor, sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_input, m_nb_branchs*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_N, sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_w, m_width*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_w_old, m_width*sizeof(float) ) );
	
	//copy
	HANDLE_ERROR( hipMemcpy( dev_nb_branchs, &m_nb_input, sizeof(float), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_input, input, m_nb_input * sizeof(float), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_N, &m_width, sizeof(float), hipMemcpyHostToDevice ) );
	
	for(int  i = 0; i<m_width; i++)
    {
		y[i] = input_layer[i].test(input);
		bias[i] = input_layer[i].m_bias;
		
    }
	
    error = m_expect - test(input);
	delete transition;
    return 0.5*error*error;
}

float Net::test(float *input){
	float *transition = new float[m_width];
	float output = 0.f;
	for(int  i = 0; i<m_width; i++){
		transition[i]  = input_layer[i].test(input);
	}
	output = output_layer->test(transition);
	delete transition;
	return output;
}

void Net::getState(){
	
	for(int i = 0; i < m_width; i++){
		cout << "Neuron nb "<< i <<" : ";
		for(int j = 0; j< m_nb_input; j++){
			cout << input_layer[i].get_weight(j) << " ";
		}
		cout << endl;
	}
	
	cout << "Output Neuron : ";
	for(int j = 0; j< m_width; j++){
		cout << output_layer->get_weight(j) << " ";
	}
	cout << endl;
	
}